#include <stdio.h>
#include <stdlib.h>
#include <random>
#include <limits>
#include <cmath>
#include <hip/hip_runtime.h>
#include "base_info.h"
#include "get_random.cu"
#include "matrix_cal.cu"

#define N (1024 * 4)

int main(){
    float gpu_elapsed_time_ms;
    hipStream_t hip_stream[2];
    hipStreamCreateWithFlags(&hip_stream[1], hipStreamNonBlocking);
    hipStreamCreateWithFlags(&hip_stream[0], hipStreamNonBlocking);

    // make event
    hipEvent_t start, stop, start1, stop1;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&start1);
    hipEventCreate(&stop1);

    // device matrix malloc
    int *d_A, *d_B, *d_C, *d_D, *d_E;
    hipMalloc((void **) &d_A, sizeof(int)*N*N);
    hipMalloc((void **) &d_B, sizeof(int)*N*N);
    hipMalloc((void **) &d_C, sizeof(int)*N*N);
    hipMalloc((void **) &d_D, sizeof(int)*N*N);
    hipMalloc((void **) &d_E, sizeof(int)*N*N);

    dim3 dimGrid(GRID_SIZE, GRID_SIZE);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    int seed;
    seed = get_seed();
    d_rand_matrix<<<1, 1>>>(seed, d_A, N);
    seed = get_seed();
    d_rand_matrix<<<1, 1>>>(seed, d_B, N);
    seed = get_seed();
    d_rand_matrix<<<1, 1>>>(seed, d_D, N);
    seed = get_seed();
    d_rand_matrix<<<1, 1>>>(seed, d_E, N);
    hipDeviceSynchronize();

    char c = 0;
    printf("set done");
    while((c = getchar()) != 'y');

    // record matrix multiple
    hipEventRecord(start, hip_stream[0]);
    hipEventRecord(start1, hip_stream[1]);
    
    
    d_mm_normal<<<dimGrid, dimBlock, 0, hip_stream[0]>>>(d_A, d_B, d_C, N);
    // d_mm_normal<<<dimGrid, dimBlock, 0, hip_stream[1]>>>(d_D, d_E, d_C, N);
    hipDeviceSynchronize();

    hipEventRecord(stop, hip_stream[0]);
    hipEventRecord(stop1, hip_stream[1]);
    hipEventSynchronize(stop);
    hipEventSynchronize(stop1);

    // calculate elapsed time
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    
    printf("normal mm : %f ms\n", gpu_elapsed_time_ms);
    hipEventElapsedTime(&gpu_elapsed_time_ms, start1, stop1);
    
    printf("normal mm : %f ms\n", gpu_elapsed_time_ms);
    
    // free
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}